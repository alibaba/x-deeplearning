/*
 * \file gemm_op.cu 
 * \brief The gemm operation on gpu implementation
 */
#include "blaze/operator/op/gemm_op.h"

namespace blaze {

REGISTER_HIP_OPERATOR(Gemm, GemmOp<HIPContext>);

}  // namespace blaze
