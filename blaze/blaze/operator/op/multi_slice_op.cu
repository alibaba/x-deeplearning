/*
 * \file multi_slice_op.cu 
 * \brief The multi slice operation
 */
#include "blaze/operator/op/multi_slice_op.h"

#include "blaze/common/common_defines.h"

namespace blaze {

template <>
void MultiSliceOp<HIPContext>::Memcpy2D(void* dst, size_t dpitch,
    const void* src, size_t spitch,
    size_t width, size_t height) const {
  hipStream_t stream = this->context_.hip_stream();
  CUDA_CHECK(hipMemcpy2DAsync(dst, dpitch, src, spitch,
        width, height, hipMemcpyDefault, stream)); 
}

template <>
bool MultiSliceOp<HIPContext>::RunOnDevice() {
  Blob* x = this->Input(0);
  TYPE_SWITCH_ON_CUDA(x->data_type(), DType, {
    MultiSliceParam<DType> param;
    Setup(&param);
    // copy data
    if (use_memcpy2d_) {
      SliceMemcpy2D<DType>();
    } else {
      MultiSliceMemcpy<DType>(); 
    }
  });
  return true;
}

REGISTER_HIP_OPERATOR(MultiSlice, MultiSliceOp<HIPContext>);

}  // namespace blaze
