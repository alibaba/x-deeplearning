/*
 * \file matmul_op.cu 
 * \brief The matmul operation
 */
#include "blaze/operator/op/matmul_op.h"

namespace blaze {

REGISTER_HIP_OPERATOR(MatMul, MatMulOp<HIPContext>);

}  // namespace blaze
