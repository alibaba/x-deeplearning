/*
 * \file flatten_op.cu
 * \brief The flatten operation
 */
#include "blaze/operator/op/flatten_op.h"

namespace blaze {

REGISTER_HIP_OPERATOR(Flatten, FlattenOp<HIPContext>);

}  // namespace blaze

