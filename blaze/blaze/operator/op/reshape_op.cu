/*
 * \file reshape_op.cu
 * \brief The reshape operation on gpu arch
 */
#include "blaze/operator/op/reshape_op.h"

namespace blaze {

REGISTER_HIP_OPERATOR(Reshape, ReshapeOp<HIPContext>);

}  // namespace blaze
