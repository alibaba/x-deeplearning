/*
 * \file fused_parallel_matmul_op.cu 
 * \brief The fused parallel matmul operation
 */
#include "blaze/operator/fused_op/fused_parallel_matmul_op.h"

namespace blaze {

REGISTER_HIP_OPERATOR(FusedParallelMatMul, FusedParallelMatMulOp<HIPContext>);

}  // namespace blaze

