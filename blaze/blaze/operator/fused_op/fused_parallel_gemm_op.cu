/*
 * \file fused_parallel_gemm_op.cu 
 * \brief The fused parallel gemm operation
 */
#include "blaze/operator/fused_op/fused_parallel_gemm_op.h"

namespace blaze {

REGISTER_HIP_OPERATOR(FusedParallelGemm, FusedParallelGemmOp<HIPContext>);

}  // namespace blaze

