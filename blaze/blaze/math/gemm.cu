#include "hip/hip_runtime.h"
/*
 * \file gemm.cu
 * \brief The gemm device kernel
 */
#include "blaze/math/gemm.h"

#include "blaze/common/common_defines.h"
#include "blaze/common/exception.h"
#include "blaze/math/float16.h"

namespace blaze {

template <>
void Gemm<float16, HIPContext>(const CBLAS_TRANSPOSE TransA,
                                const CBLAS_TRANSPOSE TransB,
                                const int M,
                                const int N,
                                const int K,
                                const float alpha,
                                const float16* A,
                                const float16* B,
                                const float beta,
                                float16* C,
                                HIPContext* ctx) {
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  int ldc = N;
  float16 alpha_h, beta_h;
  float2half(&alpha, 1, &alpha_h);
  float2half(&beta, 1, &beta_h);

  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  // We now use cublas method for smaller matrix. NOTE: Optimization on small matrix.
  CUBLAS_CHECK(hipblasHgemm(ctx->cublas_handle(),
                           cuTransB,
                           cuTransA,
                           N,
                           M,
                           K,
                           (const __half*)&alpha_h,
                           (const __half*)B,
                           ldb,
                           (const __half*)A,
                           lda,
                           (const __half*)&beta_h,
                           (__half*)C,
                           ldc)); 
}

template <>
void Gemm<float, HIPContext>(const CBLAS_TRANSPOSE TransA,
                              const CBLAS_TRANSPOSE TransB,
                              const int M,
                              const int N,
                              const int K,
                              const float alpha,
                              const float* A,
                              const float* B,
                              const float beta,
                              float* C,
                              HIPContext* ctx) {
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  int ldc = N;
  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  // We now use cublas method for smaller matrix. NOTE: Optimization on small matrix.
  CUBLAS_CHECK(hipblasSgemm(ctx->cublas_handle(),
                           cuTransB,
                           cuTransA,
                           N,
                           M,
                           K,
                           &alpha,
                           B,
                           ldb,
                           A,
                           lda,
                           &beta,
                           C,
                           ldc));
}

template <>
void Gemm<double, HIPContext>(const CBLAS_TRANSPOSE TransA,
                               const CBLAS_TRANSPOSE TransB,
                               const int M,
                               const int N,
                               const int K,
                               const float alpha,
                               const double* A,
                               const double* B,
                               const float beta,
                               double* C,
                               HIPContext* ctx) {
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  int ldc = N;
  double alpha_d = alpha, beta_d = beta;
  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  // We now use cublas method for smaller matrix. NOTE: Optimization on small matrix.
  CUBLAS_CHECK(hipblasDgemm(ctx->cublas_handle(),
                           cuTransB,
                           cuTransA,
                           N,
                           M,
                           K,
                           &alpha_d,
                           B,
                           ldb,
                           A,
                           lda,
                           &beta_d,
                           C,
                           ldc));
}

template <>
void GemmEx<float16, HIPContext>(const CBLAS_TRANSPOSE TransA,
                                  const CBLAS_TRANSPOSE TransB,
                                  const int M,
                                  const int N,
                                  const int K,
                                  const float alpha,
                                  const float16* A,
                                  const int lda,
                                  const float16* B,
                                  const int ldb,
                                  const float beta,
                                  float16* C,
                                  const int ldc,
                                  HIPContext* ctx) {
  float16 alpha_h, beta_h;
  float2half(&alpha, 1, &alpha_h);
  float2half(&beta, 1, &beta_h);
  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasHgemm(ctx->cublas_handle(),
                           cuTransB,
                           cuTransA,
                           N,
                           M,
                           K,
                           (const __half*)&alpha_h,
                           (const __half*)B,
                           ldb,
                           (const __half*)A,
                           lda,
                           (const __half*)&beta_h,
                           (__half*)C,
                           ldc));
}

template <>
void GemmEx<float, HIPContext>(const CBLAS_TRANSPOSE TransA,
                                const CBLAS_TRANSPOSE TransB,
                                const int M,
                                const int N,
                                const int K,
                                const float alpha,
                                const float* A,
                                const int lda,
                                const float* B,
                                const int ldb,
                                const float beta,
                                float* C,
                                const int ldc,
                                HIPContext* ctx) {
  float alpha_f = alpha, beta_f = beta;
  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(ctx->cublas_handle(),
                           cuTransB,
                           cuTransA,
                           N,
                           M,
                           K,
                           &alpha_f,
                           B,
                           ldb,
                           A,
                           lda,
                           &beta_f,
                           C,
                           ldc));
}

template <>
void GemmEx<double, HIPContext>(const CBLAS_TRANSPOSE TransA,
                                 const CBLAS_TRANSPOSE TransB,
                                 const int M,
                                 const int N,
                                 const int K,
                                 const float alpha,
                                 const double* A,
                                 const int lda,
                                 const double* B,
                                 const int ldb,
                                 const float beta,
                                 double* C,
                                 const int ldc,
                                 HIPContext* ctx) {
  double alpha_d = alpha, beta_d = beta;
  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(ctx->cublas_handle(),
                           cuTransB,
                           cuTransA,
                           N,
                           M,
                           K,
                           &alpha_d,
                           B,
                           ldb,
                           A,
                           lda,
                           &beta_d,
                           C,
                           ldc));
}

template <>
void GemmStridedBatched<float16, HIPContext>(const CBLAS_TRANSPOSE TransA,
                                              const CBLAS_TRANSPOSE TransB,
                                              const int M,
                                              const int N,
                                              const int K,
                                              const float alpha,
                                              const float16* A,
                                              const long long int stride_a,
                                              const float16* B,
                                              const long long int stride_b,
                                              const float beta,
                                              float16* C,
                                              const long long int stride_c,
                                              int batch_count,
                                              HIPContext* ctx) {
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  int ldc = N;
  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  
  float16 alpha_h, beta_h;
  float2half(&alpha, 1, &alpha_h);
  float2half(&beta, 1, &beta_h);

  // We now use cublas method for smaller matrix. NOTE: Optimization on small matrix.
  CUBLAS_CHECK(hipblasHgemmStridedBatched(ctx->cublas_handle(),
                                         cuTransB,
                                         cuTransA,
                                         N,
                                         M,
                                         K,
                                         (const __half*)&alpha_h,
                                         (const __half*)B,
                                         ldb,
                                         stride_b,
                                         (const __half*)A,
                                         lda,
                                         stride_a,
                                         (const __half*)&beta_h,
                                         (__half*)C,
                                         ldc,
                                         stride_c,
                                         batch_count));
}

template <>
void GemmStridedBatched<float, HIPContext>(const CBLAS_TRANSPOSE TransA,
                                            const CBLAS_TRANSPOSE TransB,
                                            const int M,
                                            const int N,
                                            const int K,
                                            const float alpha,
                                            const float* A,
                                            const long long int stride_a,
                                            const float* B,
                                            const long long int stride_b,
                                            const float beta,
                                            float* C,
                                            const long long int stride_c,
                                            int batch_count,
                                            HIPContext* ctx) {
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  int ldc = N;
  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  // We now use cublas method for smaller matrix. NOTE: Optimization on small matrix.
  CUBLAS_CHECK(hipblasSgemmStridedBatched(ctx->cublas_handle(),
                                         cuTransB,
                                         cuTransA,
                                         N,
                                         M,
                                         K,
                                         &alpha,
                                         B,
                                         ldb,
                                         stride_b,
                                         A,
                                         lda,
                                         stride_a,
                                         &beta,
                                         C,
                                         ldc,
                                         stride_c,
                                         batch_count));
}

template <>
void GemmStridedBatched<double, HIPContext>(const CBLAS_TRANSPOSE TransA,
                                             const CBLAS_TRANSPOSE TransB,
                                             const int M,
                                             const int N,
                                             const int K,
                                             const float alpha,
                                             const double* A,
                                             const long long int stride_a,
                                             const double* B,
                                             const long long int stride_b,
                                             const float beta,
                                             double* C,
                                             const long long int stride_c,
                                             int batch_count,
                                             HIPContext* ctx) {
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  int ldc = N;
  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  double alpha_d = alpha, beta_d = beta;
  // We now use cublas method for smaller matrix. NOTE: Optimization on small matrix.
  CUBLAS_CHECK(hipblasDgemmStridedBatched(ctx->cublas_handle(),
                                         cuTransB,
                                         cuTransA,
                                         N,
                                         M,
                                         K,
                                         &alpha_d,
                                         B,
                                         ldb,
                                         stride_b,
                                         A,
                                         lda,
                                         stride_a,
                                         &beta_d,
                                         C,
                                         ldc,
                                         stride_c,
                                         batch_count));
}

template <>
void GemmBatched<float16, HIPContext>(const CBLAS_TRANSPOSE TransA,
                                       const CBLAS_TRANSPOSE TransB,
                                       const int M,
                                       const int N,
                                       const int K,
                                       const float alpha,
                                       const float16* A_array[],
                                       const float16* B_array[],
                                       const float beta,
                                       float16* C_array[],
                                       int batch_count,
                                       HIPContext* ctx) {
#if CUDA_VERSION >= 9000  // Only CUDA9.0 Can support
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  int ldc = N;
  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  
  float16 alpha_h, beta_h;
  float2half(&alpha, 1, &alpha_h);
  float2half(&beta, 1, &beta_h);

  // We now use cublas method for smaller matrix. NOTE: Optimization on small matrix.
  CUBLAS_CHECK(hipblasHgemmBatched(ctx->cublas_handle(),
                                  cuTransB,
                                  cuTransA,
                                  N,
                                  M,
                                  K,
                                  (const __half*)&alpha_h,
                                  (const __half**)B_array,
                                  ldb,
                                  (const __half**)A_array,
                                  lda,
                                  (const __half*)&beta_h,
                                  (__half**)C_array,
                                  ldc,
                                  batch_count));
#else
  BLAZE_THROW("Not supported, CUDA_VERSION < 9000");
#endif
}

template <>
void GemmBatched<float, HIPContext>(const CBLAS_TRANSPOSE TransA,
                                     const CBLAS_TRANSPOSE TransB,
                                     const int M,
                                     const int N,
                                     const int K,
                                     const float alpha,
                                     const float* A_array[],
                                     const float* B_array[],
                                     const float beta,
                                     float* C_array[],
                                     int batch_count,
                                     HIPContext* ctx) {
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  int ldc = N;
  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  // We now use cublas method for smaller matrix. NOTE: Optimization on small matrix.
  CUBLAS_CHECK(hipblasSgemmBatched(ctx->cublas_handle(),
                                  cuTransB,
                                  cuTransA,
                                  N,
                                  M,
                                  K,
                                  &alpha,
                                  B_array,
                                  ldb,
                                  A_array,
                                  lda,
                                  &beta,
                                  C_array,
                                  ldc,
                                  batch_count));
}

template <>
void GemmBatched<double, HIPContext>(const CBLAS_TRANSPOSE TransA,
                                      const CBLAS_TRANSPOSE TransB,
                                      const int M,
                                      const int N,
                                      const int K,
                                      const float alpha,
                                      const double* A_array[],
                                      const double* B_array[],
                                      const float beta,
                                      double* C_array[],
                                      int batch_count,
                                      HIPContext* ctx) {
  double alpha_d = alpha, beta_d = beta;
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  int ldc = N;
  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  // We now use cublas method for smaller matrix. NOTE: Optimization on small matrix.
  CUBLAS_CHECK(hipblasDgemmBatched(ctx->cublas_handle(),
                                  cuTransB,
                                  cuTransA,
                                  N,
                                  M,
                                  K,
                                  &alpha_d,
                                  B_array,
                                  ldb,
                                  A_array,
                                  lda,
                                  &beta_d,
                                  C_array,
                                  ldc,
                                  batch_count));
}


template <>
void Gemv<float16, HIPContext>(const CBLAS_TRANSPOSE TransA,
                                const int M,
                                const int N,
                                const float alpha,
                                const float16* A,
                                const float16* x,
                                const float beta,
                                float16* y,
                                HIPContext* ctx) {
  BLAZE_THROW("Not implemented!");
}

template <>
void Gemv<float, HIPContext>(const CBLAS_TRANSPOSE TransA,
                              const int M,
                              const int N,
                              const float alpha,
                              const float* A,
                              const float* x,
                              const float beta,
                              float* y,
                              HIPContext* ctx) {
  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(ctx->cublas_handle(),
                           cuTransA,
                           N,
                           M,
                           &alpha,
                           A,
                           N,
                           x,
                           1,
                           &beta,
                           y,
                           1));
}

template <>
void Gemv<double, HIPContext>(const CBLAS_TRANSPOSE TransA,
                               const int M,
                               const int N,
                               const float alpha,
                               const double* A,
                               const double* x,
                               const float beta,
                               double* y,
                               HIPContext* ctx) {
  hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  double alpha_d = alpha, beta_d = beta;
  CUBLAS_CHECK(hipblasDgemv(ctx->cublas_handle(),
                           cuTransA,
                           N,
                           M,
                           &alpha_d,
                           A,
                           N,
                           x,
                           1,
                           &beta_d,
                           y,
                           1));
}

}  // namespace blaze

